#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
// #include "hip/hip_runtime.h"
// #include ""
#include "pgmUtility.h"
#include "pgmUtilityGPU.h"
#include "pgmProcess.h"

void usage();

int main(int argc, char *argv[]){

    // int p1[] = {1, 1};
    // int p2[] = {0, 0};x
    // double distance;
    // distance =  distanceSquared( p1, p2 );
    // printf("THIS THE ONE: %f", distance);
    
    FILE * fp = NULL;
    FILE * out = NULL;
    FILE * outGPU = NULL; 

    char ** header = (char**) malloc( sizeof(char *) * rowsInHeader);
    int i;
    int * pixels = NULL;
    int * pixelsGPU = NULL;
    for(i = 0; i < 4; i++){
        header[i] = (char *) malloc (sizeof(char) * maxSizeHeadRow);
    }
    int numRows, numCols;

    int p1y = 0;
    int p1x = 0;
    int p2y = 0;
    int p2x = 0;

    int m, n, l, x, ch;
    int edgeWidth, circleCenterRow, circleCenterCol, radius;
    char originalImageName[100], newImageFileName[100];
    if(argc != 5 && argc != 7 && argc != 8)
    {
                usage();
        return 1;
        }
    else
    {            
        l = strlen( argv[1] );
        if(l != 2){
            usage();
            return 1;
        }
        ch = (int)argv[1][1];
        if(ch < 97)
            ch = ch + 32;
        switch( ch )
        {
            case 'c':  
                if(argc != 7){
                    usage();
                    break;
                }
                circleCenterRow = atoi(argv[2]);
                circleCenterCol = atoi(argv[3]);
                radius = atoi(argv[4]);
                strcpy(originalImageName, argv[5]);
                strcpy(newImageFileName, argv[6]);

                fp = fopen(originalImageName, "r");
                if(fp == NULL){
                    usage();
                    return 1;
                }
                out = fopen(newImageFileName, "w");
                if(out == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }
                outGPU = fopen(strcat(newImageFileName, "GPU"), "w");
                if(outGPU == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }


                pixels = pgmRead(header, &numRows, &numCols, fp);
                // printArr(pixels, numRows, numCols);
                // int p1[] = {1, 1};
                // int p2[] = {0, 0};
                // double distance;
                // distance =  distanceSquared( p1, p2 );
                // printf("THIS THE ONE: %f", distance);
                // pixelsGPU = ( int * ) malloc( ( numRows ) * ( numCols) * sizeof( int ) );
                // memcpy(pixelsGPU, pixels, ( numRows ) * ( numCols) * sizeof( int ) );
                
                // CPU
                pgmDrawCircle(pixels, numRows, numCols, circleCenterRow, circleCenterCol, radius, header );
                printArr(pixels, numRows, numCols);
                // pgmWrite((const char **)header, (const int *)pixels, numRows, numCols, out );  
                pixels = pgmRead(header, &numRows, &numCols, fp);

                // GPU
                pgmDrawCircleGPU(pixels, numRows, numCols, circleCenterRow, circleCenterCol, radius, header );
                printArr(pixels, numRows, numCols);
                // pgmWrite((const char **)header, (const int *)pixelsGPU, numRows, numCols, outGPU );  

                break;
            case 'e':  
                if(argc != 5){
                    usage();
                    break;
                }
                edgeWidth = atoi(argv[2]);
                strcpy(originalImageName, argv[3]);
                strcpy(newImageFileName, argv[4]);
                fp = fopen(originalImageName, "r");
                if(fp == NULL){
                    usage();
                    return 1;
                }
                out = fopen(newImageFileName, "w");
                if(out == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }
                outGPU = fopen(strcat(newImageFileName, "GPU"), "w");
                if(outGPU == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }

                pixels = pgmRead(header, &numRows, &numCols, fp);
                pixelsGPU = pixels;

                // CPU
                pgmDrawEdge(pixels, numRows, numCols, edgeWidth, header);
                pgmWrite((const char **)header, (const int *)pixels, numRows, numCols, out );
                
                // GPU
                // pgmDrawEdgeGPU(pixelsGPU, numRows, numCols, edgeWidth, header);
                // pgmWrite((const char **)header, (const int *)pixelsGPU, numRows, numCols, outGPU );
                
                break;

            case 'l':  
                if(argc != 8){
                    usage();
                    break;
                }
                p1y = atoi(argv[2]);
                p1x = atoi(argv[3]);

                p2y = atoi(argv[4]);
                p2x = atoi(argv[5]);


                strcpy(originalImageName, argv[6]);
                strcpy(newImageFileName, argv[7]);

                fp = fopen(originalImageName, "r");
                if(fp == NULL){
                    usage();
                    return 1;
                }
                out = fopen(newImageFileName, "w");
                if(out == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }
                outGPU = fopen(strcat(newImageFileName, "GPU"), "w");
                if(outGPU == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }

                pixels = pgmRead(header, &numRows, &numCols, fp);
                pixelsGPU = pixels;

                // CPU
                pgmDrawLine(pixels, numRows, numCols, header, p1y, p1x, p2y, p2x);
                pgmWrite((const char **)header, (const int *)pixels, numRows, numCols, out );
                
                // GPU
                // pgmDrawLineGPU(pixelsGPU, numRows, numCols, header, p1y, p1x, p2y, p2x);
                // pgmWrite((const char **)header, (const int *)pixelsGPU, numRows, numCols, outGPU );
                
                break;
        }      
    }

    // Not needed for 1D representation
    // i = 0;
    // for(;i < numRows; i++)
    //     free(pixels[i]);
    free(pixels);
    free(pixelsGPU);
    i = 0;
    for(;i < rowsInHeader; i++)
        free(header[i]);
    free(header);
    if(out != NULL)
        fclose(out);
    if(fp != NULL)
        fclose(fp);

    m = 0;
    n = 0;
    x = 0;
    printf("m: %d, n: %d, x: %d\n", m, n, x);
    return 0;
}

void usage()
{
        printf("Usage:\n    -e edgeWidth  oldImageFile  newImageFile\n    -c circleCenterRow circleCenterCol radius  oldImageFile  newImageFile\n    -l  p1row  p1col  p2row  p2col  oldImageFile  newImageFile\n");

}
