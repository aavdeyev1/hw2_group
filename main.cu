#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
// #include "hip/hip_runtime.h"
// #include ""
#include "pgmUtility.h"
#include "pgmUtilityGPU.h"
#include "pgmProcess.h"

void usage();

int main(int argc, char *argv[]){

    
    FILE * fp = NULL;
    FILE * out = NULL;
    FILE * outGPU = NULL; 

    char ** header = (char**) malloc( sizeof(char *) * rowsInHeader);
    int i;
    int * pixels = NULL;
    int * pixelsGPU = NULL;
    for(i = 0; i < 4; i++){
        header[i] = (char *) malloc (sizeof(char) * maxSizeHeadRow);
    }
    int numRows, numCols;

    int p1y = 0;
    int p1x = 0;
    int p2y = 0;
    int p2x = 0;

    int m, n, l, x, ch;
    int edgeWidth, circleCenterRow, circleCenterCol, radius;
    char newImageFileName[100], originalImageName[100], newImageFileNameGPU[100];
    if(argc != 5 && argc != 7 && argc != 8)
    {
                usage();
        return 1;
        }
    else
    {            
        l = strlen( argv[1] );
        if(l != 2){
            usage();
            return 1;
        }
        ch = (int)argv[1][1];
        if(ch < 97)
            ch = ch + 32;
        switch( ch )
        {
            case 'c':  
                if(argc != 7){
                    usage();
                    break;
                }
                circleCenterRow = atoi(argv[2]);
                circleCenterCol = atoi(argv[3]);
                radius = atoi(argv[4]);

                // Get filenames from command line args
                strcpy(originalImageName, argv[5]);
                strcpy(newImageFileName, argv[6]);

                // make GPU filename without .pgm file extension
                memcpy(newImageFileNameGPU, &newImageFileName, (strlen(newImageFileName) - 4 )*sizeof(char));
                    
                fp = fopen(originalImageName, "r");
                if(fp == NULL){
                    usage();
                    return 1;
                }
                out = fopen(newImageFileName, "w");
                if(out == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }
                outGPU = fopen(strcat(newImageFileNameGPU, "GPU.pgm"), "w");
                if(outGPU == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }


                pixels = pgmRead(header, &numRows, &numCols, fp);
                pixelsGPU = ( int * ) malloc(numCols*numRows*sizeof(int));
                memcpy(pixelsGPU, pixels, numCols*numRows*sizeof(int));
                
                // CPU
                pgmDrawCircle(pixels, numRows, numCols, circleCenterRow, circleCenterCol, radius, header );
                pgmWrite((const char **)header, (const int *)pixels, numRows, numCols, out );  

                // GPU
                pgmDrawCircleGPU(pixelsGPU, numRows, numCols, circleCenterRow, circleCenterCol, radius, header );
                pgmWrite((const char **)header, (const int *)pixelsGPU, numRows, numCols, outGPU );  

                break;
            case 'e':  
                if(argc != 5){
                    usage();
                    break;
                }
                edgeWidth = atoi(argv[2]);
                // Get filenames from command line args
                strcpy(originalImageName, argv[5]);
                strcpy(newImageFileName, argv[6]);

                // make GPU filename without .pgm file extension
                memcpy(newImageFileNameGPU, &newImageFileName, (strlen(newImageFileName) - 4 )*sizeof(char));
                    
                fp = fopen(originalImageName, "r");
                if(fp == NULL){
                    usage();
                    return 1;
                }
                out = fopen(newImageFileName, "w");
                if(out == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }
                outGPU = fopen(strcat(newImageFileNameGPU, "GPU.pgm"), "w");
                if(outGPU == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }


                pixels = pgmRead(header, &numRows, &numCols, fp);
                pixelsGPU = ( int * ) malloc(numCols*numRows*sizeof(int));
                memcpy(pixelsGPU, pixels, numCols*numRows*sizeof(int));

                // CPU
                pgmDrawEdge(pixels, numRows, numCols, edgeWidth, header);
                pgmWrite((const char **)header, (const int *)pixels, numRows, numCols, out );
                
                // GPU
                pgmDrawEdgeGPU(pixelsGPU, numRows, numCols, edgeWidth, header);
                pgmWrite((const char **)header, (const int *)pixelsGPU, numRows, numCols, outGPU );
                
                break;

            case 'l':  
                if(argc != 8){
                    printf("HERE: %d", argc);
                    usage();
                    break;
                }
                p1y = atoi(argv[2]);
                p1x = atoi(argv[3]);

                p2y = atoi(argv[4]);
                p2x = atoi(argv[5]);


                // Get filenames from command line args
                strcpy(originalImageName, argv[5]);
                strcpy(newImageFileName, argv[6]);

                // make GPU filename without .pgm file extension
                memcpy(newImageFileNameGPU, &newImageFileName, (strlen(newImageFileName) - 4 )*sizeof(char));
                    
                fp = fopen(originalImageName, "r");
                if(fp == NULL){
                    printf("HERE 1: %d", argc);
                    usage();
                    return 1;
                }
                out = fopen(newImageFileName, "w");
                if(out == NULL){
                    printf("HERE 2: %d", argc);
                    usage();
                    fclose(fp);
                    return 1;
                }
                outGPU = fopen(strcat(newImageFileNameGPU, "GPU.pgm"), "w");
                if(outGPU == NULL){
                    printf("HERE 3: %d", argc);
                    usage();
                    fclose(fp);
                    return 1;
                }


                pixels = pgmRead(header, &numRows, &numCols, fp);
                pixelsGPU = ( int * ) malloc(numCols*numRows*sizeof(int));
                memcpy(pixelsGPU, pixels, numCols*numRows*sizeof(int));
                
                // CPU
                pgmDrawLine(pixels, numRows, numCols, header, p1y, p1x, p2y, p2x);
                pgmWrite((const char **)header, (const int *)pixels, numRows, numCols, out );
                
                // GPU
                pgmDrawLineGPU(pixelsGPU, numRows, numCols, header, p1y, p1x, p2y, p2x);
                pgmWrite((const char **)header, (const int *)pixelsGPU, numRows, numCols, outGPU );
                
                break;
        }      
    }

    free(pixels);
    free(pixelsGPU);
    i = 0;
    for(;i < rowsInHeader; i++)
        free(header[i]);
    free(header);
    if(out != NULL)
        fclose(out);
    if(fp != NULL)
        fclose(fp);

    m = 0;
    n = 0;
    x = 0;
    printf("m: %d, n: %d, x: %d\n", m, n, x);
    return 0;
}

void usage()
{
        printf("Usage:\n    -e edgeWidth  oldImageFile  newImageFile\n    -c circleCenterRow circleCenterCol radius  oldImageFile  newImageFile\n    -l  p1row  p1col  p2row  p2col  oldImageFile  newImageFile\n");

}
