#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>

#include <fstream>
#include <iostream>

#include "pgmUtility.h"
#include "pgmUtilityGPU.h"
#include "pgmProcess.h"

void usage();

int main(int argc, char *argv[]){
    // CPU processed files are stored as the filename given in the args
    // GPU processed files are stored under the same filename but prefixed with GPU_
    std::ofstream myfile;
    clock_t startCPU;
    double cpuStart;
    clock_t endCPU;
    double cpuEnd;

    clock_t startGPU;
    double gpuStart;
    clock_t endGPU;
    double gpuEnd;

    long double diffCPU;
    long double diffGPU;


    FILE * fp = NULL;
    FILE * out = NULL;
    FILE * outGPU = NULL; 



    char ** header = (char**) malloc( sizeof(char *) * rowsInHeader);
    int i;
    int * pixels = NULL;
    int * pixelsGPU = NULL;
    for(i = 0; i < 4; i++){
        header[i] = (char *) malloc (sizeof(char) * maxSizeHeadRow);
    }
    int numRows, numCols;

    int p1y = 0;
    int p1x = 0;
    int p2y = 0;
    int p2x = 0;

    int l, ch;
    int edgeWidth, circleCenterRow, circleCenterCol, radius;
    char newImageFileName[100], originalImageName[100];
    char gpu[] = "GPU_";
    if(argc != 5 && argc != 7 && argc != 8)
    {
                usage();
        return 1;
        }
    else
    {            
        l = strlen( argv[1] );
        if(l != 2){
            usage();
            return 1;
        }
        ch = (int)argv[1][1];
        if(ch < 97)
            ch = ch + 32;
        switch( ch )
        {
            case 'c':  
                if(argc != 7){
                    usage();
                    break;
                }
                circleCenterRow = atoi(argv[2]);
                circleCenterCol = atoi(argv[3]);
                radius = atoi(argv[4]);

                // Get filenames from command line args
                strcpy(originalImageName, argv[5]);
                strcpy(newImageFileName, argv[6]);
                    
                fp = fopen(originalImageName, "r");
                if(fp == NULL){
                    usage();
                    return 1;
                }
                out = fopen(newImageFileName, "w");
                if(out == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }
                outGPU = fopen(strcat(gpu, newImageFileName), "w");
                if(outGPU == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }


                pixels = pgmRead(header, &numRows, &numCols, fp);
                pixelsGPU = ( int * ) malloc(numCols*numRows*sizeof(int));
                memcpy(pixelsGPU, pixels, numCols*numRows*sizeof(int));
                
                // CPU
                startCPU = clock();
                cpuStart = (double) startCPU/CLOCKS_PER_SEC;
                pgmDrawCircle(pixels, numRows, numCols, circleCenterRow, circleCenterCol, radius, header );
                pgmWrite((const char **)header, (const int *)pixels, numRows, numCols, out );  
                endCPU = clock();
                cpuEnd = (double) endCPU/CLOCKS_PER_SEC;
                // GPU
                startGPU = clock();
                gpuStart = (double) startGPU/CLOCKS_PER_SEC;               
                pgmDrawCircleGPU(pixelsGPU, numRows, numCols, circleCenterRow, circleCenterCol, radius, header );
                pgmWrite((const char **)header, (const int *)pixelsGPU, numRows, numCols, outGPU );  
                endGPU = clock();
                gpuEnd = (double) endGPU/CLOCKS_PER_SEC;


                diffCPU = cpuEnd - cpuStart;
                diffGPU = gpuEnd - gpuStart;

                //then send them all to the an output file
                myfile.open ("TimeResults");
                myfile << "CPU Process Time: " << diffCPU << "\n";
                myfile << "GPU Process Time: " << diffGPU << "\n\n";
                myfile.close();                                

                break;
            case 'e':
                if(argc != 5){
                    usage();
                    break;
                }
                edgeWidth = atoi(argv[2]);

                strcpy(originalImageName, argv[3]);
                strcpy(newImageFileName, argv[4]);

                fp = fopen(originalImageName, "r");
                if(fp == NULL){
                    usage();
                    return 1;
                }
                out = fopen(newImageFileName, "w");
                if(out == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }
                outGPU = fopen(strcat(gpu, newImageFileName), "w");
                if(outGPU == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }

                pixels = pgmRead(header, &numRows, &numCols, fp);

                pixelsGPU = ( int * ) malloc(numCols*numRows*sizeof(int));
                memcpy(pixelsGPU, pixels, numCols*numRows*sizeof(int));

                // CPU
                startCPU = clock();
                cpuStart = (double) startCPU/CLOCKS_PER_SEC;
                pgmDrawEdge(pixels, numRows, numCols, edgeWidth, header);
                pgmWrite((const char **)header, (const int *)pixels, numRows, numCols, out );
                endCPU = clock();
                cpuEnd = (double) endCPU/CLOCKS_PER_SEC;
                // GPU
                startGPU = clock();
                gpuStart = (double) startGPU/CLOCKS_PER_SEC;
                pgmDrawEdgeGPU(pixelsGPU, numRows, numCols, edgeWidth, header);
                pgmWrite((const char **)header, (const int *)pixelsGPU, numRows, numCols, outGPU );
                endGPU = clock();
                gpuEnd = (double) endGPU/CLOCKS_PER_SEC;

                diffCPU = cpuEnd - cpuStart;
                diffGPU = gpuEnd - gpuStart;

                //then send them all to the an output file
                myfile << "CPU Process Time: " << diffCPU << "\n";
                myfile << "GPU Process Time: " << diffGPU << "\n\n";
                myfile.close();                                

                break;

            case 'l':  
                if(argc != 8){
                    printf("HERE: %d", argc);
                    usage();
                    break;
                }
                p1y = atoi(argv[2]);
                p1x = atoi(argv[3]);

                p2y = atoi(argv[4]);
                p2x = atoi(argv[5]);


                // Get filenames from command line args
                strcpy(originalImageName, argv[6]);
                strcpy(newImageFileName, argv[7]);
                    
                fp = fopen(originalImageName, "r");
                if(fp == NULL){
                    usage();
                    return 1;
                }
                out = fopen(newImageFileName, "w");
                if(out == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }
                outGPU = fopen(strcat(gpu, newImageFileName), "w");
                if(outGPU == NULL){
                    usage();
                    fclose(fp);
                    return 1;
                }


                pixels = pgmRead(header, &numRows, &numCols, fp);
                pixelsGPU = ( int * ) malloc(numCols*numRows*sizeof(int));
                memcpy(pixelsGPU, pixels, numCols*numRows*sizeof(int));
                
                // CPU
                startCPU = clock();
                cpuStart = (double) startCPU/CLOCKS_PER_SEC;                
                pgmDrawLine(pixels, numRows, numCols, header, p1y, p1x, p2y, p2x);
                pgmWrite((const char **)header, (const int *)pixels, numRows, numCols, out );
                endCPU = clock();
                cpuEnd = (double) endCPU/CLOCKS_PER_SEC;               
                // GPU
                startGPU = clock();
                gpuStart = (double) startGPU/CLOCKS_PER_SEC;                
                pgmDrawLineGPU(pixelsGPU, numRows, numCols, header, p1y, p1x, p2y, p2x);
                pgmWrite((const char **)header, (const int *)pixelsGPU, numRows, numCols, outGPU );
                endGPU = clock();
                gpuEnd = (double) endGPU/CLOCKS_PER_SEC;

                diffCPU = cpuEnd - cpuStart;
                diffGPU = gpuEnd - gpuStart;

                //then send them all to the an output file
                myfile << "CPU Process Time: " << diffCPU << "\n";
                myfile << "GPU Process Time: " << diffGPU << "\n\n";
                myfile.close();                                
                break;
        }      
    }
    free(pixels);
    free(pixelsGPU);
    i = 0;
    for(;i < rowsInHeader; i++)
        free(header[i]);
    free(header);
    if(out != NULL)
        fclose(out);
    if(fp != NULL)
        fclose(fp);

    return 0;
}

void usage()
{
        printf("Usage:\n    -e edgeWidth  oldImageFile  newImageFile\n    -c circleCenterRow circleCenterCol radius  oldImageFile  newImageFile\n    -l  p1row  p1col  p2row  p2col  oldImageFile  newImageFile\n");

}
