#include "hip/hip_runtime.h"

#include "pgmProcess.h"
#include "hip/hip_runtime.h"
#include ""
/**
 *  Function Name:
 *      distance()
 *      distance() returns the Euclidean distance between two pixels. This function is executed on CUDA device
 *
 *  @param[in]  p1  coordinates of pixel one, p1[0] is for row number, p1[1] is for column number
 *  @param[in]  p2  coordinates of pixel two, p2[0] is for row number, p2[1] is for column number
 *  @return         return distance between p1 and p2
 */
__device__ float distanceSquared( int p1[], int p2[] )
{
    p1[1] = (float) p1[1];
    p1[0] = (float) p1[0];
    p2[1] = (float) p2[1];
    p2[0] = (float) p2[0];
    return (p1[0] - p2[0])*(p1[0] - p2[0]) + (p1[1] - p2[1])*( p1[1] - p2[1]);
}

//returns either 0 or 1
//0 = outside of radius; pixel will not be changed
//1 = within radius bounds; pixel will be changed to black
__global__ void calcDist(int *pixels, int numRows, int numCols, int centCol, int centRow, int radius) {
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*numCols + ix;
    
    int p1[2] = {ix, iy}; //array to hold our x and y values for given point
    int p2[2] = {centCol, centRow}; //given centerpoint will never be changed

    //use distance function to find the distance
    float disSq = distanceSquared(p1, p2);

    if ((disSq <= (float)(radius * radius )) && ix < numCols) { //if distance is within radius of center point...
        pixels[idx] = 0;
    }
    
}

__global__ void edgeKernel( int *a, int dimx, int dimy, int w )
{
	int ix   = blockIdx.x*blockDim.x + threadIdx.x;
	int iy   = blockIdx.y*blockDim.y + threadIdx.y;
	int idx = iy*dimx + ix;
	if(ix<dimx && iy<dimy)//checks if it is in bounds
		if(ix<w||iy<w||ix>=(dimx-w)||iy>=(dimy-w))//check if it is on the edge
				a[idx]  = 0;
}


__global__ void gpuLineDraw(int* array4GPU, int numRows, int numCols, int p1row, int p1col, int p2row, int p2col, float slope, float b, int tempMaxX, int tempMaxY, int tempMinX, int tempMinY, float range) {

    //calculate global X id // use this to determine if in range of the line
    int xId = threadIdx.x + blockDim.x * blockIdx.x;
    //calculate global Y id // use this to determine if in range of the line
    int yId = threadIdx.y + blockDim.y * blockIdx.y;
    //calculate global thread ID
    int globalId = xId + (yId * numCols);


    //if threads outside range of "2d" array do nothing
    if (xId < numCols && yId < numRows) {
        //if threads are within tempMinX <-> tempMaxX and tempMinY <-> tempMaxY
        if (xId <= tempMaxX && xId >= tempMinX && yId <= tempMaxY && yId >= tempMinY) {//threads within the area where the line is drawn
            float xVal = (slope * xId + b);//this displays the y value that should be filled in at this X// this is on the line
            float yVal = (float)yId;//this is the y value we are currently at may or maynot be on the line
            // if threads y value is on line change to 0
            if (yVal < xVal + range && yVal > xVal - range) {
                array4GPU[globalId] = 0;
            }
        }
    }
}