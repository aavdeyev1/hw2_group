#include "hip/hip_runtime.h"

#include "pgmProcess.h"
#include "hip/hip_runtime.h"
#include ""
/**
 *  Function Name:
 *      distance()
 *      distance() returns the Euclidean distance between two pixels. This function is executed on CUDA device
 *
 *  @param[in]  p1  coordinates of pixel one, p1[0] is for row number, p1[1] is for column number
 *  @param[in]  p2  coordinates of pixel two, p2[0] is for row number, p2[1] is for column number
 *  @return         return distance between p1 and p2
 */
__device__ float distanceSquared( int p1[], int p2[] )
{
    p1[1] = (float) p1[1];
    p1[0] = (float) p1[0];
    p2[1] = (float) p2[1];
    p2[0] = (float) p2[0];
    return (p1[0] - p2[0])*(p1[0] - p2[0]) + (p1[1] - p2[1])*( p1[1] - p2[1]);
}

//returns either 0 or 1
//0 = outside of radius; pixel will not be changed
//1 = within radius bounds; pixel will be changed to black
__global__ void calcDist(int *pixels, int numRows, int numCols, int centCol, int centRow, int radius) {
    int ix   = blockIdx.x*blockDim.x + threadIdx.x;
    int iy   = blockIdx.y*blockDim.y + threadIdx.y;
    int idx = iy*numCols + ix;
    
    int p1[2] = {ix, iy}; //array to hold our x and y values for given point
    int p2[2] = {centCol, centRow}; //given centerpoint will never be changed
    
    // float rad = (float) radius; //variable to hold a double version or radius
    //for comparisons

    //use distance function to find the distance
    float disSq = distanceSquared(p1, p2);
    // pixels[idx] = (int)ceil(disSq);
    // pixels[idx] = idx;

    if ((disSq <= (float)(rad * rad)) || ix < numCols) { //if distance is within radius of center point...
        pixels[idx] = 0;
    }
    
}