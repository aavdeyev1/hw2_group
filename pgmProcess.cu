
#include "pgmProcess.h"
#include "hip/hip_runtime.h"
#include ""
/**
 *  Function Name:
 *      distance()
 *      distance() returns the Euclidean distance between two pixels. This function is executed on CUDA device
 *
 *  @param[in]  p1  coordinates of pixel one, p1[0] is for row number, p1[1] is for column number
 *  @param[in]  p2  coordinates of pixel two, p2[0] is for row number, p2[1] is for column number
 *  @return         return distance between p1 and p2
 */
// __device__ float distanceSquared( int p1[], int p2[] )
// {
//     p1[1] = (float) p1[1];
//     p1[0] = (float) p1[0];
//     p2[1] = (float) p2[1];
//     p2[0] = (float) p2[0];
//     return (p1[0] - p2[0])*(p1[0] - p2[0]) + (p1[1] - p2[1])*( p1[1] - p2[1]);
// }