#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtility.h"
#include "pgmUtilityGPU.h"
#include "pgmProcess.h"

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.


//---------------------------------------------------------------------------
//
int pgmDrawCircleGPU( int *pixelsGPU, int numRows, int numCols, int centerRow,
    int centerCol, int radius, char **header )
{

    int *d_pixels=0; // device pointers
    int bytes = numRows * numCols * sizeof( int);

  //   h_a = (int*)malloc(num_bytes);
    hipMalloc( &d_pixels, bytes );

    if( 0==d_pixels )
    {
        printf("couldn't allocate memory\n");
        return -1;
    }
    
    hipMemset( d_pixels, 0, num_bytes );
    // hipMemcpy( d_pixels, pixelsGPU, bytes, hipMemcpyHostToDevice);

    dim3 grid, block;

    block.x = 32;
    block.y = 32;
    grid.x  = ceil( (float)numCols / block.x );
    grid.y  = ceil( (float)numRows / block.y );

    // printf("grid: %d, %d\nblock: %d, %d\n", grid.x, grid.y, block.x, block.y);
    
    
  //   // Use kernel to fill d_a array
  calcDist<<<grid, block>>>(pixelsGPU, numRows, numCols, centerCol, centerRow, radius);
  hipMemcpy( pixelsGPU, d_pixels, bytes, hipMemcpyDeviceToHost );
  printArr(pixelsGPU, numRows, numCols);
  //   strcpy(somestr, " kernel ");
  // // boo = calcDist(j, i, centerCol, centerRow, radius);

  //         //if our 'boolean' is 'true'...
  //   pixels[(i * numCols + j)] = 0;

  //   free( h_a );
    hipFree( d_pixels );

  //   free(somestr);

  return 0; // :)

}

//---------------------------------------------------------------------------
int pgmDrawEdgeGPU( int *pixels, int numRows, int numCols, int edgeWidth, char **header )
{
  return 0;
}

//---------------------------------------------------------------------------

int pgmDrawLineGPU( int *pixels, int numRows, int numCols, char **header,
  int p1row, int p1col, int p2row, int p2col )
{

  return 0;
}
