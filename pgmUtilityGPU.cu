#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtilityGPU.h"

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.


//---------------------------------------------------------------------------
//
int pgmDrawCircleGPU( int *pixels, int numRows, int numCols, int centerRow,
    int centerCol, int radius, char **header )
{

    int *d_pixels=0; // device pointers
    size_t bytes = numRows * numCols * sizeof( int );

  //   h_a = (int*)malloc(num_bytes);
    hipMalloc( (void*)&d_pixels, bytes );

    if( 0==h_a || 0==d_a )
    {
        printf("couldn't allocate memory\n");
        return 1;
    }

    hipMemset( d_pixels, 0, bytes );

  //   dim3 grid, block;
  //   block.x = 3;
  //   block.y = 4;
  //   grid.x  = ceil( (float)dimx / block.x );
  //   grid.y  = ceil( (float)dimy / block.y );
  //   char *somestr = (char *)malloc(9*sizeof(char));
    
    
  //   // Use kernel to fill d_a array
  //   kernel<<<grid, block>>>( d_a, dimx, dimy );
    hipMemcpy( h_a, d_a, num_bytes, hipMemcpyDeviceToHost );
  //   strcpy(somestr, " kernel ");
  // // boo = calcDist(j, i, centerCol, centerRow, radius);

  //         //if our 'boolean' is 'true'...
  //   pixels[(i * numCols + j)] = 0;

  //   free( h_a );
    hipFree( d_a );

  //   free(somestr);

  return 0; // :)

}

//---------------------------------------------------------------------------
int pgmDrawEdgeGPU( int *pixels, int numRows, int numCols, int edgeWidth, char **header )
{

}

//---------------------------------------------------------------------------

int pgmDrawLineGPU( int *pixels, int numRows, int numCols, char **header,
  int p1row, int p1col, int p2row, int p2col )
{

}
