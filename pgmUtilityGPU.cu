#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtility.h"
#include "pgmUtilityGPU.h"
#include "pgmProcess.h"

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.


//---------------------------------------------------------------------------
//
int pgmDrawCircleGPU( int *pixelsGPU, int numRows, int numCols, int centerRow,
    int centerCol, int radius, char **header )
{

    int *h_pixels=0, *d_pixels=0; // host/device pointers
    int bytes = numRows * numCols * sizeof( int);

    h_pixels = (int*)malloc(bytes);
    hipMalloc( &d_pixels, bytes );

    if( 0==d_pixels )
    {
        printf("couldn't allocate memory\n");
        return -1;
    }
    
    hipMemcpy( d_pixels, pixelsGPU, bytes, hipMemcpyHostToDevice);

    dim3 grid, block;

    block.x = 32;
    block.y = 32;
    grid.x  = ceil( (float)numCols / block.x );
    grid.y  = ceil( (float)numRows / block.y );

    printf("\ngrid: %d, %d\nblock: %d, %d\n", bytes, grid.x, grid.y, block.x, block.y);
    
    calcDist<<<grid, block>>>(d_pixels, numRows, numCols, centerCol, centerRow, radius);
    hipMemcpy( h_pixels, d_pixels, bytes, hipMemcpyDeviceToHost );
    memcpy(pixelsGPU, h_pixels, bytes);

    free( h_pixels );
    hipFree( d_pixels );

  return 0; // :)

}

//---------------------------------------------------------------------------
int pgmDrawEdgeGPU( int *pixels, int numRows, int numCols, int edgeWidth, char **header )
{
  return 0;
}

//---------------------------------------------------------------------------

int pgmDrawLineGPU( int *pixels, int numRows, int numCols, char **header,
  int p1row, int p1col, int p2row, int p2col )
{

  return 0;
}
