#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtility.h"
#include "pgmUtilityGPU.h"
#include "pgmProcess.h"

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.


//---------------------------------------------------------------------------
//
int pgmDrawCircleGPU( int *pixelsGPU, int numRows, int numCols, int centerRow,
    int centerCol, int radius, char **header )
{

    int *h_pixels=0, *d_pixels=0; // device pointers
    int bytes = numRows * numCols * sizeof( int);

    h_pixels = (int*)malloc(bytes);
    hipMalloc( &d_pixels, bytes );

    if( 0==d_pixels )
    {
        printf("couldn't allocate memory\n");
        return -1;
    }
    
    // hipMemset( d_pixels, 0, bytes );
    hipMemcpy( d_pixels, pixelsGPU, bytes, hipMemcpyHostToDevice);

    dim3 grid, block;

    block.x = 32;
    block.y = 32;
    grid.x  = ceil( (float)numCols / block.x );
    grid.y  = ceil( (float)numRows / block.y );

    printf("%d, grid: %d, %d\nblock: %d, %d\n", bytes, grid.x, grid.y, block.x, block.y);
    
    
    //   // Use kernel to fill d_a array
    calcDist<<<grid, block>>>(d_pixels, numRows, numCols, centerCol, centerRow, radius);
    hipMemcpy( h_pixels, d_pixels, bytes, hipMemcpyDeviceToHost );
    memcpy(pixelsGPU, h_pixels, bytes);
    //   strcpy(somestr, " kernel ");
    // // boo = calcDist(j, i, centerCol, centerRow, radius);

    //         //if our 'boolean' is 'true'...
    //   pixels[(i * numCols + j)] = 0;

    free( h_pixels );
    hipFree( d_pixels );

    //   free(somestr);

  return 0; // :)

}

//---------------------------------------------------------------------------
int pgmDrawEdgeGPU( int *pixels, int numRows, int numCols, int edgeWidth, char **header )
{
  return 0;
}

//---------------------------------------------------------------------------

int pgmDrawLineGPU( int *pixels, int numRows, int numCols, char **header,
  int p1row, int p1col, int p2row, int p2col )
{

  return 0;
}
