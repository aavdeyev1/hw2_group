#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtilityGPU.h"

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.


//---------------------------------------------------------------------------
//
int pgmDrawCircleGPU( int *pixels, int numRows, int numCols, int centerRow,
    int centerCol, int radius, char **header )
{

}

//---------------------------------------------------------------------------
int pgmDrawEdgeGPU( int *pixels, int numRows, int numCols, int edgeWidth, char **header )
{
    int *d_pixels=0; // device pointers
    int bytes = numRows * numCols * sizeof( int );

  //   h_a = (int*)malloc(num_bytes);
    hipMalloc( (void**)&d_pixels, bytes );

    if( 0==d_pixels )
    {
        printf("couldn't allocate memory\n");
        return -1;
    }
    hipMemcpy( d_pixels, pixels, bytes, hipMemcpyHostToDevice);

    dim3 grid, block;

    block.x = 32;
    block.y = 32;
    grid.x  = ceil( (float)numCols / block.x );
    grid.y  = ceil( (float)numRows / block.y );
    
    
  edgeKernel<<<grid, block>>>( d_pixels , numCols, numRows, edgeWidth );
  hipMemcpy( pixels, d_pixels, bytes, hipMemcpyDeviceToHost );
  //printArr(pixels, "edge ", numRows, numCols);
  hipFree( d_pixels );
  return 1;
}

//---------------------------------------------------------------------------

int pgmDrawLineGPU( int *pixels, int numRows, int numCols, char **header,
  int p1row, int p1col, int p2row, int p2col )
{

}
