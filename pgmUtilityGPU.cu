#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtilityGPU.h"

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.


//---------------------------------------------------------------------------
//
int pgmDrawCircleGPU( int *pixels, int numRows, int numCols, int centerRow,
    int centerCol, int radius, char **header )
{

    int *d_pixels=0; // device pointers
    int bytes = numRows * numCols * sizeof( int );

  //   h_a = (int*)malloc(num_bytes);
    hipMalloc( (void*)&d_pixels, bytes );

    if( 0==d_pixels )
    {
        printf("couldn't allocate memory\n");
        return 1;
    }

    hipMemset( d_pixels, 0, bytes );

  //   dim3 grid, block;
  //   block.x = 3;
  //   block.y = 4;
  //   grid.x  = ceil( (float)dimx / block.x );
  //   grid.y  = ceil( (float)dimy / block.y );
  //   char *somestr = (char *)malloc(9*sizeof(char));
    
    
  //   // Use kernel to fill d_a array
  //   kernel<<<grid, block>>>( d_a, dimx, dimy );
    hipMemcpy( pixels, d_pixels, bytes, hipMemcpyDeviceToHost );
  //   strcpy(somestr, " kernel ");
  // // boo = calcDist(j, i, centerCol, centerRow, radius);

  //         //if our 'boolean' is 'true'...
  //   pixels[(i * numCols + j)] = 0;

  //   free( h_a );
    hipFree( d_a );

  //   free(somestr);

  return 0; // :)

}

//---------------------------------------------------------------------------
int pgmDrawEdgeGPU( int *pixels, int numRows, int numCols, int edgeWidth, char **header )
{
  return 0;
}

//---------------------------------------------------------------------------

int pgmDrawLineGPU( int *pixels, int numRows, int numCols, char **header,
  int p1row, int p1col, int p2row, int p2col )
{

  return 0;
}
