#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pgmUtility.h"
#include "pgmUtilityGPU.h"
#include "pgmProcess.h"
#include "hip/hip_runtime.h"
#include ""

// Implement or define each function prototypes listed in pgmUtility.h file.
// NOTE: Please follow the instructions stated in the write-up regarding the interface of the functions.
// NOTE: You might have to change the name of this file into pgmUtility.cu if needed.


//---------------------------------------------------------------------------
//
int pgmDrawCircleGPU( int *pixelsGPU, int numRows, int numCols, int centerRow,
    int centerCol, int radius, char **header )
{

    int *h_pixels=0, *d_pixels=0; // host/device pointers
    int bytes = numRows * numCols * sizeof( int);

    h_pixels = (int*)malloc(bytes);
    hipMalloc( &d_pixels, bytes );

    if( 0==d_pixels )
    {
        printf("couldn't allocate memory\n");
        return -1;
    }
    
    hipMemcpy( d_pixels, pixelsGPU, bytes, hipMemcpyHostToDevice);

    dim3 grid, block;

    block.x = 32;
    block.y = 32;
    grid.x  = ceil( (float)numCols / block.x );
    grid.y  = ceil( (float)numRows / block.y );

    printf("\ngrid: %d, %d\nblock: %d, %d\n", grid.x, grid.y, block.x, block.y);
    
    calcDist<<<grid, block>>>(d_pixels, numRows, numCols, centerCol, centerRow, radius);
    hipMemcpy( h_pixels, d_pixels, bytes, hipMemcpyDeviceToHost );
    memcpy(pixelsGPU, h_pixels, bytes);

    free( h_pixels );
    hipFree( d_pixels );

  return 0; // :)

}

//---------------------------------------------------------------------------
int pgmDrawEdgeGPU( int *pixelsGPU, int numRows, int numCols, int edgeWidth, char **header )
{
  //   int *d_pixels=0; // device pointers
  //   int bytes = numRows * numCols * sizeof( int );

  // //   h_a = (int*)malloc(num_bytes);
  //   hipMalloc( (void**)&d_pixels, bytes );

  //   if( 0==d_pixels )
  //   {
  //       printf("couldn't allocate memory\n");
  //       return -1;
  //   }
  //   hipMemcpy( d_pixels, pixels, bytes, hipMemcpyHostToDevice);

  //   dim3 grid, block;

  //   block.x = 32;
  //   block.y = 32;
  //   grid.x  = ceil( (float)numCols / block.x );
  //   grid.y  = ceil( (float)numRows / block.y );

  
  int *h_pixels=0, *d_pixels=0; // host/device pointers
    int bytes = numRows * numCols * sizeof( int);

    h_pixels = (int*)malloc(bytes);
    hipMalloc( &d_pixels, bytes );

    if( 0==d_pixels )
    {
        printf("couldn't allocate memory\n");
        return -1;
    }
    
    hipMemcpy( d_pixels, pixelsGPU, bytes, hipMemcpyHostToDevice);

    dim3 grid, block;

    block.x = 32;
    block.y = 32;
    grid.x  = ceil( (float)numCols / block.x );
    grid.y  = ceil( (float)numRows / block.y );

    printf("\ngrid: %d, %d\nblock: %d, %d\n", grid.x, grid.y, block.x, block.y);
    
    
  edgeKernel<<<grid, block>>>( d_pixels , numCols, numRows, edgeWidth );

  hipMemcpy( h_pixels, d_pixels, bytes, hipMemcpyDeviceToHost );
  memcpy(pixelsGPU, h_pixels, bytes);

  free( h_pixels );
  hipFree( d_pixels );

  // hipMemcpy( pixels, d_pixels, bytes, hipMemcpyDeviceToHost );
  // //printArr(pixels, "edge ", numRows, numCols);
  // hipFree( d_pixels );
  return 1;
}

//---------------------------------------------------------------------------

int pgmDrawLineGPU( int *pixels, int numRows, int numCols, char **header,  int p1row, int p1col, int p2row, int p2col )
{

    float slope;
    float b;
    int tempMaxX, tempMaxY;
    int tempMinX, tempMinY;

    slope = ((float)(p2row - p1row)) / ((float)(p2col - p1col));
    b = p1row - slope * p1col;

    float range = slope / 2;
    if (slope > -1 || slope < 1) {
        range = .51;
    }

    if (p2row < p1row) {
        tempMinY = p2row;
        tempMaxY = p1row;
    }
    else {
        tempMinY = p1row;
        tempMaxY = p2row;
    }
    if (p2col < p1col) {
        tempMinX = p2col;
        tempMaxX = p1col;
    }
    else {
        tempMinX = p1col;
        tempMaxX = p2col;
    }



    //allocate memory on GPU
    int* array4GPU=0;
    size_t bytes = (sizeof(int) * numCols * numRows);
    hipMalloc(&array4GPU, bytes);
    //copy memory from CPU - > GPU
    hipMemcpy(array4GPU, pixels, bytes, hipMemcpyHostToDevice);
    //calculate gridsize and block size
    dim3 grid, block;

    block.x = 32;
    block.y = 32;
    block.z = 1;
    grid.x  = ceil( (float)numCols / block.x );
    grid.y  = ceil( (float)numRows / block.y );
    grid.z = 1;
    // dim3 blockDim = (32, 32, 1);// (x,y,z)// 1024 threads a block
    // dim3 gridDim = (ceil(numCols / 32), ceil(numRows / 32), 1);//(x,y,z)
    //call the kernel

    gpuLineDraw <<< grid, block >>> (array4GPU, numRows, numCols, p1row, p1col, p2row, p2col, slope, b, tempMaxX, tempMaxY, tempMinX, tempMinY, range);

    //copy memory from GPU - > CPU
    hipMemcpy(pixels, array4GPU, bytes, hipMemcpyDeviceToHost);
    //free memory from gpu
    hipFree(&array4GPU);


  return 0;
}

